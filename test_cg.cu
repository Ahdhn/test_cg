#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdint.h>
#include <cstdint>
#include <numeric>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;


//********************** CUDA_ERROR
inline void HandleError(hipError_t err, const char *file, int line) {
	//Error handling micro, wrap it around function whenever possible
	if (err != hipSuccess) {
		printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);

#ifdef _WIN32
		system("pause");
#else
		exit(EXIT_FAILURE);
#endif
	}
}
#define CUDA_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//******************************************************************************

//********************** Get Cuda Arch
__global__ void get_cude_arch_k(int*d_arch){

#if defined(__CUDA_ARCH__)
	*d_arch = __CUDA_ARCH__;
#else
	*d_arch = 0;
#endif
}
inline int cuda_arch(){
	int*d_arch = 0;
	CUDA_ERROR(hipMalloc((void**)&d_arch, sizeof(int)));
	get_cude_arch_k << < 1, 1 >> >(d_arch);
	int h_arch = 0;
	CUDA_ERROR(hipMemcpy(&h_arch, d_arch, sizeof(int), hipMemcpyDeviceToHost));
	hipFree(d_arch);
	return h_arch;
}
//******************************************************************************


//********************** testing cg kernel 
__global__ void testing_cg_grid_sync(const uint32_t num_elements,
	uint32_t *d_arr){
	
	uint32_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid < num_elements){

		uint32_t my_element = d_arr[tid];
				
		cg::grid_group barrier = cg::this_grid();
		//cg::thread_block barrier = cg::this_thread_block();

		//wait for all reads 
		barrier.sync();		

		uint32_t tar_id = num_elements - tid - 1;

		d_arr[tar_id] = my_element;
	}	
}
//******************************************************************************


//********************** execute  
void execute_test(const int sm_count){

	//check cuda arc	
	//int my_cuda_arch = cuda_arch(); 
	//if (my_cuda_arch< 600){
	//	printf("\n ERROR: with compute capability < 600, cooperative groups"
	//		"can not sync across blocks.");
	//	exit(EXIT_FAILURE);
	//}else{
	//	printf("\n __CUDA_ARCH__ = %d", my_cuda_arch);
	//}

	//host array 
	const uint32_t arr_size = 1 << 20;
	uint32_t* h_arr = (uint32_t*)malloc(arr_size * sizeof(uint32_t));
	//with with sequential numbers
	std::iota(h_arr, h_arr + arr_size, 0);

	//device array 
	uint32_t* d_arr;
	CUDA_ERROR(hipMalloc((void**)&d_arr, arr_size*sizeof(uint32_t)));
	CUDA_ERROR(hipMemcpy(d_arr, h_arr, arr_size*sizeof(uint32_t), 
		hipMemcpyHostToDevice));

	//launch config
	const int threads = 512;

	int num_blocks_per_sm = 0;
	CUDA_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm,
		(void*)testing_cg_grid_sync, threads, 0));

	dim3 grid_dim(sm_count * num_blocks_per_sm, 1, 1), block_dim(threads, 1, 1);

	printf("\n Launching %d blcoks, each containing %d threads", grid_dim.x,
		block_dim.x);
	
	//argument passed to the kernel 	
	void *kernel_args[] = {		
		(void *)&arr_size,
	    (void *)&d_arr,};


	//finally launch the kernel 
	hipLaunchCooperativeKernel((void*)testing_cg_grid_sync,
		grid_dim, block_dim, kernel_args);


	//make sure everything went okay
	CUDA_ERROR(hipGetLastError());
	CUDA_ERROR(hipDeviceSynchronize());
	

	//get results on the host 
	CUDA_ERROR(hipMemcpy(h_arr, d_arr, arr_size*sizeof(uint32_t),
		hipMemcpyDeviceToHost));

	//validate 
	for (uint32_t i = 0; i < arr_size; i++){
		if (h_arr[i] != arr_size - i - 1){
			printf("\n Result mismatch in h_arr[%u] = %u\n", i, h_arr[i]);
			exit(EXIT_FAILURE);
		}
	}	
}
//******************************************************************************

int main(int argc, char**argv) {

	//set to Titan V
	uint32_t device_id = 0;	
	hipSetDevice(device_id);

	//get sm count 
	hipDeviceProp_t devProp;	
	CUDA_ERROR(hipGetDeviceProperties(&devProp, device_id));
	int sm_count = devProp.multiProcessorCount;
	
	//execute 
	execute_test(sm_count);

	printf("\n Mission accomplished \n");
	return 0;
}
