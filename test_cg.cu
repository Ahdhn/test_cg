#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdint.h>
#include <cstdint>
#include <numeric>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;


//********************** CUDA_ERROR
inline void HandleError(hipError_t err, const char *file, int line) {
	//Error handling micro, wrap it around function whenever possible
	if (err != hipSuccess) {
		printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);

#ifdef _WIN32
		system("pause");
#else
		exit(EXIT_FAILURE);
#endif
	}
}
#define CUDA_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//******************************************************************************


//********************** testing cg kernel 
__global__ void testing_cg_grid_sync(const uint32_t num_elements,
	uint32_t *d_arr){
	
	uint32_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid < num_elements){

		uint32_t my_element = d_arr[tid];
				
		//to sync across the whole grid 
		cg::grid_group barrier = cg::this_grid();

		//to sync within a single block 
		//cg::thread_block barrier = cg::this_thread_block();

		//wait for all reads 
		barrier.sync();		

		uint32_t tar_id = num_elements - tid - 1;

		d_arr[tar_id] = my_element;
	}
	return;
}
//******************************************************************************


//********************** execute  
void execute_test(const int sm_count){
	
	//host array 
	//const uint32_t arr_size = 1 << 20; //1M 
	const uint32_t arr_size = 1680*80;
	uint32_t* h_arr = (uint32_t*)malloc(arr_size * sizeof(uint32_t));
	//with with sequential numbers
	std::iota(h_arr, h_arr + arr_size, 0);

	//device array 
	uint32_t* d_arr;
	CUDA_ERROR(hipMalloc((void**)&d_arr, arr_size*sizeof(uint32_t)));
	CUDA_ERROR(hipMemcpy(d_arr, h_arr, arr_size*sizeof(uint32_t), 
		hipMemcpyHostToDevice));

	//launch config
	const int threads = 80;

	//following the same steps done in conjugateGradientMultiBlockCG.cu 
	//cuda sample to launch kernel that sync across grid 
	//https://github.com/NVIDIA/cuda-samples/blob/master/Samples/conjugateGradientMultiBlockCG/conjugateGradientMultiBlockCG.cu#L436

	int num_blocks_per_sm = 0;
	CUDA_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm,
		(void*)testing_cg_grid_sync, threads, 0));

	dim3 grid_dim(sm_count * num_blocks_per_sm, 1, 1), block_dim(threads, 1, 1);

	printf("\n Launching %d blcoks, each containing %d threads \n", grid_dim.x,
		block_dim.x);

	if(arr_size > grid_dim.x*block_dim.x){
         printf("\n The grid size (numBlocks*numThreads) is less than array size.\n");
         printf("This will result into mismatch error (incorrect output erro)\n");
         exit(EXIT_FAILURE);
    }

    if((int(grid_dim.x*block_dim.x) - int(arr_size)) / threads > 0 ){
    	printf("\n At least one block might not see the sync barrier. This will (probabily) result into the code never exits.\n");
    	exit(EXIT_FAILURE);
    }
		
	//argument passed to the kernel 	
	void *kernel_args[] = {		
		(void *)&arr_size,
	    (void *)&d_arr,};


	//finally launch the kernel 
	hipLaunchCooperativeKernel((void*)testing_cg_grid_sync,
		grid_dim, block_dim, kernel_args);


	//make sure everything went okay
	CUDA_ERROR(hipGetLastError());
	CUDA_ERROR(hipDeviceSynchronize());
	

	//get results on the host 
	CUDA_ERROR(hipMemcpy(h_arr, d_arr, arr_size*sizeof(uint32_t),
		hipMemcpyDeviceToHost));

	//validate 
	for (uint32_t i = 0; i < arr_size; i++){
		if (h_arr[i] != arr_size - i - 1){
			printf("\n Result mismatch in h_arr[%u] = %u\n", i, h_arr[i]);
			exit(EXIT_FAILURE);
		}
	}	
}
//******************************************************************************

int main(int argc, char**argv) {

	//set to Titan V
	uint32_t device_id = 0;	
	hipSetDevice(device_id);

	//get sm count 
	hipDeviceProp_t devProp;	
	CUDA_ERROR(hipGetDeviceProperties(&devProp, device_id));
	int sm_count = devProp.multiProcessorCount;
	
	//execute 
	execute_test(sm_count);

	printf("\n Mission accomplished \n");
	return 0;
}
